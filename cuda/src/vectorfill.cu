#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

__global__ void cuda_VectorFill(unsigned int *a,
																int sz) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	a[idx] = idx;
}

void cu_vectorFill()
{
	// Some stuff we need to perform timings
	int sz = 1000000;

	thrust::device_vector<unsigned int> a(sz);
	thrust::fill(a.begin(), a.end(), 0);

	unsigned int *a_devPtr = thrust::raw_pointer_cast(&a[0]);

	int threadsPerBlock = 1024;
	int nBlocks = sz / threadsPerBlock + 1;
	if((sz % threadsPerBlock) > 0) {
		nBlocks += 1;
	}
	
	cuda_VectorFill<<< threadsPerBlock, nBlocks >>>(a_devPtr, sz);

	hipDeviceSynchronize();

	std::cout << "Filled vector: ";
	thrust::copy(a.begin(), a.end(), std::ostream_iterator<unsigned int>(std::cout, " "));
	std::cout << "\nVector fill of " << sz << " elements on the GPU finished successfully!" << "\n";
}

